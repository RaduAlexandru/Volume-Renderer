#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>;
using namespace::std;

__global__ void kernel(int a, int b, int* result_dev)
{
	// ...
	printf("hello from kernel\n");
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	*result_dev = a + b;
	printf("result_dev from kernel es %d\n", *result_dev);
}

extern "C" void launch_kernel()
{


	int* result_dev;
	int* result_host;

	result_host = (int*)malloc(sizeof(int));
	hipMalloc(&result_dev, sizeof(int));


	std::cout << "hola" << endl;
	kernel << < 1, 2 >> > (50, 50, result_dev);

	hipMemcpy(result_host, result_dev, sizeof(int), hipMemcpyDeviceToHost);
	cout << *result_host << endl;


	printf("RUN CUDA KERNEL\n");
	//kernel << <1, 1 >> >();
}




